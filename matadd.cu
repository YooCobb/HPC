
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 10
#define N 100
#define A(i,j) A[i*N+j]
#define B(i,j) B[i*N+j]
#define C(i,j) C[i*N+j]

//CUDA kernel
__device__ float add(float a, float b)
{
   int c;
   c = a + b;
   return c;
}


__global__ void matAdd(float *A, float *B, float *C)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   int j = blockDim.y*blockIdx.y+threadIdx.y;

   C(i,j) = add(A(i,j), B(i,j));
}

int main()
{
   float A[N][N], B[N][N], C[N][N], C2[N][N], *A_d, *B_d, *C_d;
   int i,j;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

   for (i=0; i<N; i++) {
      for(j=0;j<N;j++) {
         A[i][j] = i*2;
         B[i][j] = N-i;
         C2[i][j] = A[i][j] + B[i][j]; // C2 is used to check the resultsa
      }
   }
   //Allocate array on device
   hipMalloc((void**) &A_d, sizeof(float)*N*N);
   hipMalloc((void**) &B_d, sizeof(float)*N*N);
   hipMalloc((void**) &C_d, sizeof(float)*N*N);

   //Copy array from host memory to device memory
   hipMemcpy(A_d, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(float)*N*N, hipMemcpyHostToDevice);
   matAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   //Copy the result back
   hipMemcpy(C, C_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);

   //Check the results
   for (i=0;i<N;i++)
      for(j=0;j<N;j++)
         if (C[i][j] != C2[i][j]) printf("%d %f %f\n", i, C[i][j], C2[i][j]);

   //Free memory on the device
   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);
}

